#include "hip/hip_runtime.h"
// ;-*- mode: c;-*-
// Matrix multiply device code
#include <assert.h>
#include <math.h>
#include "../src/utils.h"
#include "../src/types.h"
#include "mytypes.h"

using namespace std;

#include <stdio.h>

#define Cy 128
#define Cx 128
#define Cc 16

#define ILP 8

#define globA(x, y) __ldg(&A[x*N + y])
#define globB(x, y) __ldg(&B[x*N + y])
#define globC(x, y) C[x*N + y]

__global__ void matMul(int N, _DOUBLE_ * __restrict C, _DOUBLE_ * __restrict A, _DOUBLE_ * __restrict B){
	//local shared storage
	__shared__ _DOUBLE_ As[Cy][Cc];
	__shared__ _DOUBLE_ Bs[Cc][Cx];

	const int tx = threadIdx.x;
	const int bx = ILP*blockIdx.x;

	const int ty = threadIdx.y;
	const int by = ILP*blockIdx.y;

	const int J = bx*blockDim.x + tx;
	const int I = by*blockDim.y + ty;

	_DOUBLE_ Cij[ILP][ILP] = {0};

	#pragma unroll
	for (int kk = 0; kk < (N+Cc-1)/Cc; kk++){
		#pragma unroll
		for (int load = 0; load < ILP; load ++){
				if (I + 16*load < N && kk*Cc + tx < N) As[ty + 16*load][tx] = globA((I + 16*load), (kk*Cc + tx)); else As[ty + 16*load][tx] = 0;

				if (kk*Cc + ty < N && J + 16*load < N) Bs[ty][tx + 16*load] = globB((kk*Cc + ty), (J + 16*load)); else Bs[ty][tx + 16*load] = 0;
		}	
		
		__syncthreads();
        #pragma unroll
		for (int k = 0; k < Cc; k++){
            #pragma unroll
            for (int i = 0; i < ILP; i++){
                #pragma unroll
                for (int j = 0; j < ILP; j++){
                    Cij[i][j] += As[ty + 16*j][k]*Bs[k][tx + 16*i]; 
                }
            }
        }
		__syncthreads();
	}

    #pragma unroll
    for (int i = 0; i < ILP; i++){
        #pragma unroll
        for (int j = 0; j < ILP; j++){
            if (I + 16*j < N && J + 16*i < N)
            globC((I + 16*j), (J + 16*i)) =  Cij[i][j]; 
        }
    }


}
