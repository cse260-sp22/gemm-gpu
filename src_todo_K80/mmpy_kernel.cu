#include "hip/hip_runtime.h"
// ;-*- mode: c;-*-
// Matrix multiply device code
#include <assert.h>
#include <math.h>
#include "../src/utils.h"
#include "../src/types.h"
#include "mytypes.h"

using namespace std;

#include <stdio.h>

#define Cy 16
#define Cx 16
#define Cc 16

#define ILP 2

#define globA(x, y) __ldg(&A[x*N + y])
#define globB(x, y) __ldg(&B[x*N + y])
#define globC(x, y) C[x*N + y]

__global__ void matMul(int N, _DOUBLE_ * __restrict C, _DOUBLE_ * __restrict A, _DOUBLE_ * __restrict B){

	//local shared storage
	__shared__ _DOUBLE_ As[Cy][Cc];
	__shared__ _DOUBLE_ Bs[Cc][Cx];

	const int tx = threadIdx.x;
	const int bx = ILP*blockIdx.x;

	const int ty = threadIdx.y;
	const int by = ILP*blockIdx.y;

	const int J = bx*blockDim.x + tx;
	const int I = by*blockDim.y + ty;

	_DOUBLE_ Cij[ILP][ILP] = {0};

	#pragma unroll
	for (int kk = 0; kk < (N+Cc-1)/Cc; kk++){
		#pragma unroll
		for (int load = 0; load < ILP; load ++){
				if (I + 32*load < N && kk*Cc + tx < N) As[ty + 32*load][tx] = globA((I + 32*load), (kk*Cc + tx)); else As[ty + 32*load][tx] = 0;

				if (kk*Cc + ty < N && J + 32*load < N) Bs[ty][tx + 32*load] = globB((kk*Cc + ty), (J + 32*load)); else Bs[ty][tx + 32*load] = 0;
		}	
		
		__syncthreads();
        #pragma unroll
		for (int k = 0; k < Cc; k++){
            #pragma unroll
            for (int i = 0; i < ILP; i++){
                #pragma unroll
                for (int j = 0; j < ILP; j++){
                    Cij[i][j] += As[ty + 32*j][k]*Bs[k][tx + 32*i]; 
                }
            }
        }
		__syncthreads();
	}

    #pragma unroll
    for (int i = 0; i < ILP; i++){
        #pragma unroll
        for (int j = 0; j < ILP; j++){
            if (I + 8*j < N && J + 8*i < N)
                globC((I + 8*j), (J + 8*i)) =  Cij[i][j]; 
        }
    }


}
