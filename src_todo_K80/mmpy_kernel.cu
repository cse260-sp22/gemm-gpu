#include "hip/hip_runtime.h"
// ;-*- mode: c;-*-
// Matrix multiply device code
#include <assert.h>
#include <math.h>
#include "../src/utils.h"
#include "../src/types.h"
#include "mytypes.h"

using namespace std;

#include <stdio.h>

// #define Cy 128
// #define Cx 128
// #define Cc 16

// #define ILP 8

#define globA(x, y) __ldg(&A[x*N + y])
#define globB(x, y) __ldg(&B[x*N + y])
#define globC(x, y) C[x*N + y]

__global__ void matMul(int N, _DOUBLE_ * __restrict C, _DOUBLE_ * __restrict A, _DOUBLE_ * __restrict B){
// 	ILP 2x2, 32, 32, 32:
// 256 - 400
// 512 - 424
// 1024 - 440
// 2048 - 442.9
const int Cy = 128;
const int Cx = 128;
const int Cc = 16;

const int creg_idx = 8;

int ILP = 8;

	if (N < 300) {
// 		Cy = 32;
// Cx = 32;
// Cc = 16;

ILP = 2;
	}

	//local shared storage
	__shared__ _DOUBLE_ As[Cy][Cc];
	__shared__ _DOUBLE_ Bs[Cc][Cx];

	const int tx = threadIdx.x;
	const int bx = ILP*blockIdx.x;

	const int ty = threadIdx.y;
	const int by = ILP*blockIdx.y;

	const int J = bx*blockDim.x + tx;
	const int I = by*blockDim.y + ty;

	_DOUBLE_ Cij[creg_idx][creg_idx] = {0};

	#pragma unroll
	for (int kk = 0; kk < (N+Cc-1)/Cc; kk++){
		#pragma unroll
		for (int load = 0; load < ILP; load ++){
				if (I + 16*load < N && kk*Cc + tx < N) As[ty + 16*load][tx] = globA((I + 16*load), (kk*Cc + tx)); else As[ty + 16*load][tx] = 0;
				if (I + 16*load < N && kk*Cc + tx + 16 < N) As[ty + 16*load][tx + 16] = globA((I + 16*load), (kk*Cc + tx + 16)); else As[ty + 16*load][tx + 16] = 0;

				if (kk*Cc + ty < N && J + 16*load < N) Bs[ty][tx + 16*load] = globB((kk*Cc + ty), (J + 16*load)); else Bs[ty][tx + 16*load] = 0;
				if (kk*Cc + ty + 16 < N && J + 16*load < N) Bs[ty + 16][tx + 16*load] = globB((kk*Cc + ty + 16), (J + 16*load)); else Bs[ty + 16][tx + 16*load] = 0;
		}	
		
		__syncthreads();
        #pragma unroll
		for (int k = 0; k < Cc; k++){
            #pragma unroll
            for (int i = 0; i < ILP; i++){
                #pragma unroll
                for (int j = 0; j < ILP; j++){
                    Cij[i][j] += As[ty + 16*j][k]*Bs[k][tx + 16*i]; 
                }
            }
        }
		__syncthreads();
	}

    #pragma unroll
    for (int i = 0; i < ILP; i++){
        #pragma unroll
        for (int j = 0; j < ILP; j++){
            if (I + 16*j < N && J + 16*i < N)
                globC((I + 16*j), (J + 16*i)) =  Cij[i][j]; 
        }
    }


}
