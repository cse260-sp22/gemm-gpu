#include "hip/hip_runtime.h"
// ;-*- mode: c;-*-
// Matrix multiply device code
#include <assert.h>
#include <math.h>
#include "../src/utils.h"
#include "../src/types.h"
#include "mytypes.h"

using namespace std;

#include <stdio.h>

#define globA(x, y) A[x*N + y]
#define globB(x, y) B[x*N + y]
#define globC(x, y) C[x*N + y]

__global__ void matMul(int N, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B){

	//local shared storage
	__shared__ double As[TW][TW];
	__shared__ double Bs[TW][TW];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int by = blockIdx.y;

	int J = bx*TW + tx;
	int I = by*TW + ty;

	double Cij = 0;

	for (int kk = 0; kk < (N+TW-1)/TW; kk++){
	
		if (I < N && (kk*TW + tx) < N) As[ty][tx] = globA(I, (kk*TW + tx));
		else As[ty][tx] = 0;

		if ((kk*TW + ty) < N && J < N) Bs[ty][tx] = globB((kk*TW+ty), J);
		else Bs[ty][tx] = 0;
		
		__syncthreads();

		for (int k = 0; k < TW; k++)
			Cij += As[ty][k] * Bs[k][tx];
		__syncthreads();
	}

	if (I < N && J < N) globC(I, J) = Cij;

}
