#include "hip/hip_runtime.h"
// ;-*- mode: c;-*-
// Matrix multiply device code
#include <assert.h>
#include <math.h>
#include "../src/utils.h"
#include "../src/types.h"
#include "mytypes.h"

using namespace std;

#include <stdio.h>

#define globA(x, y) A[x*N + y]
#define globB(x, y) A[x*N + y]

//__global__ void matMul(int N, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B)
//{
//
//    int I = blockIdx.y * blockDim.y + threadIdx.y;
//    int J = blockIdx.x * blockDim.x + threadIdx.x;
//
//    if ((I < N) && (J < N))
//    {
//        _DOUBLE_ _c = 0;
//        for (unsigned int k = 0; k < N; k++)
//        {
//            _DOUBLE_ a = A[I * N + k];
//            _DOUBLE_ b = B[k * N + J];
//            _c += a * b;
//        }
//        C[I * N + J] = _c;
//    }
//}

__global__ void matMul(int N, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B){

	//local shared storage
	__shared__ double As[TW][TW];
	__shared__ double Bs[TW][TW];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int by = blockIdx.y;

	int J = bx*TW + tx;
	int I = by*TW + ty;

	double Cij = 0;

	for (int kk = 0; kk < (N+TW-1)/TW; kk++){
	
		if (I < N && (kk*TW + tx) < N){
			As[ty][tx] = A[I*N + kk*TW + tx];
		}
		else As[ty][tx] = 0;

		if ((kk*TW + ty) < N && J < N){
			Bs[ty][tx] = B[(kk*TW+ty)*N + J];
		}
		else Bs[ty][tx] = 0;
		
		__syncthreads();

		for (int k = 0; k < TW; k++)
			Cij += As[ty][k] * Bs[k][tx];
		__syncthreads();
	}

	if (I < N && J < N)
		C[I*N + J] = Cij;

}
